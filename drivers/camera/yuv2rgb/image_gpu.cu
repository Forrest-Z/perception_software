#include "hip/hip_runtime.h"
/*!
* \brief This file defines image gpu class for objects parsing and transferring
* \attention Copyright Geely car Co.Ltd
* \attention Please refer to COPYRIGHT.txt for complete terms of copyright notice. */

#include "image_gpu.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

inline __device__ __host__ float clamp(float f, float a, float b)
{
    return fmaxf(a, fminf(f, b));
}


/* From RGB to YUV

   Y = 0.299R + 0.587G + 0.114B
   U = 0.492 (B-Y)
   V = 0.877 (R-Y)

   It can also be represented as:

   Y =  0.299R + 0.587G + 0.114B
   U = -0.147R - 0.289G + 0.436B
   V =  0.615R - 0.515G - 0.100B

   From YUV to RGB

   R = Y + 1.140V
   G = Y - 0.395U - 0.581V
   B = Y + 2.032U
 */

struct __align__(2) uchar6
{
   uchar3 a0, a1;
};
static __host__ __device__ __forceinline__ uchar6 make_uchar6(uchar3 a0, uchar3 a1)
{
   uchar6 val = {a0, a1};
   return val;
}

//__global__ void yuyvToRgb( uchar4* src, int srcAlignedWidth, uchar6* dst, int dstAlignedWidth, int width, int height )
__global__ void yuyvToRgb(uint8_t* yuyv, uint8_t* rgb, int srcAlignedWidth, int dstAlignedWidth, int width, int height )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;


	if( x >= srcAlignedWidth || y >= height )
		return;

	//if(y* srcAlignedWidth +x >= 1920/1080*2/4)
	//{
	    //printf("there is a boundary %d\n", y* srcAlignedWidth +x);
	    //return;
	//}

	//const uchar4 macroPx = ((uchar4*)yuyv)[y * 1920/2 + x];// = src[y * srcAlignedWidth + x];
	const uchar4 macroPx = ((uchar4*)yuyv)[y * 960 + x];// = src[y * srcAlignedWidth + x];
	

	// Y0 is the brightness of pixel 0, Y1 the brightness of pixel 1.
	// U0 and V0 is the color of both pixels.
	// UYVY [ U0 | Y0 | V0 | Y1 ] 
	// YUYV [ Y0 | U0 | Y1 | V0 ]
	const float y0 = macroPx.x;
	const float y1 = macroPx.z; 
	const float u = (macroPx.y) - 128.0f;
	const float v = (macroPx.w) - 128.0f;

	const float3 px0 = make_float3( y0 + 1.4065f * v,
				        y0 - 0.3455f * u - 0.7169f * v,
					y0 + 1.7790f * u);

	const float3 px1 = make_float3( y1 + 1.4065f * v,
					y1 - 0.3455f * u - 0.7169f * v,
					y1 + 1.7790f * u);

	((uchar6*)rgb)[y * 1920/2 + x] = make_uchar6( make_uchar3(clamp(px0.x, 0.0f, 255.0f), 
							       clamp(px0.y, 0.0f, 255.0f),
							       clamp(px0.z, 0.0f, 255.0f)),
                                                    make_uchar3(clamp(px1.x, 0.0f, 255.0f), 
							       clamp(px1.y, 0.0f, 255.0f),
							       clamp(px1.z, 0.0f, 255.0f)));
     
} 


void CudaYUYVToRGB(const int32_t& cols, const int32_t& rows, uint8_t* cuda_yuyv, uint8_t* cuda_rgb, uint8_t* yuyv, uint8_t* rgbb_image)
{
	hipError_t cuda_status;

//added begin
#if 0
	 //GPU
	float time_GPU; 
	hipEvent_t start_GPU, stop_GPU; 
	//CPU
	double time_cpu; 
	clock_t start_cpu, stop_cpu;
	start_cpu = clock();
#endif
//added end

	cuda_status = hipSetDevice(0);
	if(cuda_status != hipSuccess)
	{
	    printf( "Device does not support cuda\n");
	    return;// cuda_status;
	}
	//printf( "step1: hipSetDevice ok \n");

	cuda_status = hipMemcpy(cuda_yuyv, yuyv, cols * rows * 2, hipMemcpyHostToDevice);
	if(cuda_status != hipSuccess)
	{
	    printf( "Could not copy data to gpu\n");
	    return;// cuda_status;
	}

	dim3 block(8,8);
//	dim3 grid(cols/2/8, rows/8);
	dim3 grid((cols/2+7)/8, rows/8);
	
    const int srcAlignedWidth = cols*rows*2 / sizeof(uchar4);	// normally would be uchar2, but we're doubling up pixels
	const int dstAlignedWidth = cols*rows*3 / sizeof(uchar6);	// normally would be uchar4 ^^^

	//printf("yuyvToRgba %zu %zu %i %i %i %i %i\n", width, height, (int)formatUYVY, srcAlignedWidth, dstAlignedWidth, grid.x, grid.y);
	//printf("yuyvToRgba %i %i %i %i\n", srcAlignedWidth, dstAlignedWidth, grid.x, grid.y);

	//yuyvToRgb<<<grid, block>>>((uchar4*)cuda_yuyv, srcAlignedWidth, (uchar6*)cuda_rgb, dstAlignedWidth, cols, rows);
//added begin
#if 0
 	//Event 
	hipEventCreate(&start_GPU); 
	hipEventCreate(&stop_GPU);
	//
	hipEventRecord(start_GPU, 0); 
#endif
//added end
	yuyvToRgb<<<grid, block>>>(cuda_yuyv, cuda_rgb, srcAlignedWidth, dstAlignedWidth, cols, rows);

    cuda_status = hipGetLastError();
        
    if(cuda_status != hipSuccess)
	{
	    printf( "Yuyv to rgb error\n");
	    return;// cuda_status;
	}

    cuda_status = hipDeviceSynchronize();
    if(cuda_status != hipSuccess)
	{
	    printf("sychronization error\n");
	    return;// cuda_status;
	}
//added begin
#if 0
 	hipEventRecord(stop_GPU, 0); 
	hipEventSynchronize(start_GPU); 
	//
	hipEventSynchronize(stop_GPU); 
	//
	hipEventElapsedTime(&time_GPU, start_GPU, stop_GPU); 
	//
	printf("\nThe time for GPU:\t%f(ms)\n", time_GPU);
#endif
//added end

    cuda_status = hipMemcpy(rgbb_image, cuda_rgb, cols*rows*3, hipMemcpyDeviceToHost);
    if(cuda_status != hipSuccess)
	{
	    printf("Could not copy data from gpu\n");
	    return;// cuda_status;
	}
#if 0
//added begin
	stop_cpu = clock();
	hipEventDestroy(start_GPU);    //destory the event
	hipEventDestroy(stop_GPU);
//added end

	printf("The time for GPU by event:\t%fms\n", time_GPU); 
	time_cpu = (double)(stop_cpu - start_cpu) / CLOCKS_PER_SEC*1000; 
	printf("\nThe time for CPU by host:\t%fms\n", time_cpu); 
#endif
        return;// cuda_status;
}